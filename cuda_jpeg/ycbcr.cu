#include "hip/hip_runtime.h"
#include "function.h"
#include<iostream>
__global__ void rgbToYCbCr(const unsigned char *img, double *y, double *cb, double *cr, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int yIdx = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && yIdx < height) {
        int idx = yIdx * width + x;
        int rgb_idx = 3 * idx;

        // 將 unsigned char 轉成 double 來計算
        double r = (double)img[rgb_idx];
        double g = (double)img[rgb_idx + 1];
        double b = (double)img[rgb_idx + 2];

        y[idx] = 0.299 * r + 0.587 * g + 0.114 * b;
        cb[idx] = -0.168736 * r - 0.331264 * g + 0.5 * b + 128;
        cr[idx] = 0.5 * r - 0.418688 * g - 0.081312 * b + 128;
    }
}

__global__ void yCbCrToRGB(const double *y, const double *cb, const double *cr, unsigned char *img, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int yIdx = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && yIdx < height) {
        int idx = yIdx * width + x;
        int rgb_idx = 3 * idx;

        double Y = y[idx];
        double Cb = cb[idx] - 128.0;
        double Cr = cr[idx] - 128.0;

        // 計算 RGB 值
        double r = Y + 1.402 * Cr;
        double g = Y - 0.344136 * Cb - 0.714136 * Cr;
        double b = Y + 1.772 * Cb;
    
        // 加上 0.5 做四捨五入，並確保值在 0-255 範圍內
        img[rgb_idx] = (unsigned char)fmin(fmax(r, 0), 255);
        img[rgb_idx + 1] = (unsigned char)fmin(fmax(g, 0), 255);
        img[rgb_idx + 2] = (unsigned char)fmin(fmax(b, 0), 255);
    }
}

void convertRGBToYCbCr(const unsigned char *img, double **yCbCr, int width, int height) {
    double *y, *cb, *cr;
    double *d_y, *d_cb, *d_cr;
    unsigned char *d_img;
    size_t imgSize = width * height * 3 * sizeof(unsigned char);
    size_t channelSize = width * height * sizeof(double);

    // Allocate device memory
    hipMalloc((void**)&d_img, imgSize);
    hipMalloc((void**)&d_y, channelSize);
    hipMalloc((void**)&d_cb, channelSize);
    hipMalloc((void**)&d_cr, channelSize);

    // Copy image data from host to device
    hipMemcpy(d_img, img, imgSize, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 block(32, 32);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // Check max threads per block
    if (block.x * block.y > MAX_THREADS_PER_BLOCK) {
        printf("Error: block size too large\n");
        return;
    }

    // Launch kernel
    rgbToYCbCr<<<grid, block>>>(d_img, d_y, d_cb, d_cr, width, height);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf(hipGetErrorString(err));
        return;
    }
    // Allocate host memory for output channels
    y = (double*)malloc(channelSize);
    cb = (double*)malloc(channelSize);
    cr = (double*)malloc(channelSize);

    // Copy results from device to host
    hipMemcpy(y, d_y, channelSize, hipMemcpyDeviceToHost);
    hipMemcpy(cb, d_cb, channelSize, hipMemcpyDeviceToHost);
    hipMemcpy(cr, d_cr, channelSize, hipMemcpyDeviceToHost);

    yCbCr[0] = y;
    yCbCr[1] = cb;
    yCbCr[2] = cr;
    // Free device memory
    hipFree(d_img);
    hipFree(d_y);
    hipFree(d_cb);
    hipFree(d_cr);
}

void convertYCbCrToRGB(double **yCbCr, unsigned char *img, int width, int height) {
    double *d_y, *d_cb, *d_cr;
    unsigned char *d_img;
    size_t imgSize = width * height * 3 * sizeof(unsigned char);
    size_t channelSize = width * height * sizeof(double);

    // Allocate device memory
    hipMalloc((void**)&d_y, channelSize);
    hipMalloc((void**)&d_cb, channelSize);
    hipMalloc((void**)&d_cr, channelSize);
    hipMalloc((void**)&d_img, imgSize);
    // Copy YCbCr data from host to device
    hipMemcpy(d_y, yCbCr[0], channelSize, hipMemcpyHostToDevice);
    hipMemcpy(d_cb, yCbCr[1], channelSize, hipMemcpyHostToDevice);
    hipMemcpy(d_cr, yCbCr[2], channelSize, hipMemcpyHostToDevice);
    // Define block and grid sizes
    dim3 block(32, 32);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // Launch kernel
    yCbCrToRGB<<<grid, block>>>(d_y, d_cb, d_cr, d_img, width, height);
    hipDeviceSynchronize();
    // Copy results from device to host
    hipMemcpy(img, d_img, imgSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_y);
    hipFree(d_cb);
    hipFree(d_cr);
    hipFree(d_img);
}

